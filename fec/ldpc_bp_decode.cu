#include "hip/hip_runtime.h"

#include "ldpc_bp_decode.cuh"
#include "ldpc_bp_decode_kernel.cuh"
//#include "driverUtility.h"
#include "itppUtility.h"
#include "dvbUtility.h"

#include <hip/hip_runtime.h>
#include <thrust/reduce.h>
#include <thrust/device_vector.h>
#include <iostream>
using namespace std;
#if USE_TEXTURE_ADDRESS
	hipArray* arr_mcv;
	hipArray* arr_mvc;
	hipChannelFormatDesc channelDesc;
#endif
#include "helper_timer.h"
#define		TIME_STEP		6	
#define		ENABLE_TIME_TEST		0	

bool ldpc_gpu::syndrome_check_gpu( int nMulti ) 
{// 0.07 ms/1f, 
	dim3 block( SIZE_BLOCK );
	dim3 grid( (ncheck + block.x - 1) / block.x );

	syndrome_check_kernel<<< grid, block >>>( d_LLRout, m_ldpcCurrent->d_sumX2, ncheck, m_ldpcCurrent->d_V, d_synd, nvar, nMulti );

	int h_synd=0;
	hipMemcpy( &h_synd, d_synd, sizeof(int), hipMemcpyDeviceToHost );

	return h_synd == 0;   // codeword is valid
}

void ldpc_gpu::updateVariableNode_gpu() 
{
	dim3 block( SIZE_BLOCK );
	dim3 grid( (nvar + block.x - 1) / block.x );

	updateVariableNode_kernel<<< grid, block >>>( nvar, ncheck, m_ldpcCurrent->d_sumX1, m_ldpcCurrent->d_mcv, m_ldpcCurrent->d_iind, d_LLRin, d_LLRout, m_ldpcCurrent->d_mvc );
}

void ldpc_gpu::updateCheckNode_gpu()
{
	dim3 block( SIZE_BLOCK );
	dim3 grid( (ncheck + block.x - 1) / block.x );

	updateCheckNode_kernel<<< grid, block >>>(ncheck, nvar, 
		m_ldpcCurrent->d_sumX2, m_ldpcCurrent->d_mvc, m_ldpcCurrent->d_jind, m_ldpcCurrent->d_logexp_table, 
		m_ldpcCurrent->Dint1, m_ldpcCurrent->Dint2, m_ldpcCurrent->Dint3,
		QLLR_MAX, m_ldpcCurrent->d_mcv );	// Shared not faster
}

void ldpc_gpu::initializeMVC_gpu( )
{
	dim3 block( 256 );
	dim3 grid( (nvar + block.x - 1) / block.x );

	initializeMVC_kernel<<< grid, block >>>( nvar, m_ldpcCurrent->d_sumX1, d_LLRin, m_ldpcCurrent->d_mvc, m_ldpcCurrent->nmaxX1, 1 );
}

int ldpc_gpu::bp_decode(int *LLRin, int *LLRout,
	bool psc /*= true*/,			//!< check syndrom after each iteration
	int max_iters /*= 50*/ )		//!< Maximum number of iterations
{
	hipMemcpy( d_LLRin, LLRin, nvar * sizeof(int), hipMemcpyHostToDevice );

  // initial step
	initializeMVC_gpu();

  bool is_valid_codeword = false;
  int iter = 0;
  do {
    iter++;
    //if (nvar >= 100000) { it_info_no_endl_debug("."); }
    // --------- Step 1: check to variable nodes ----------
	updateCheckNode_gpu();

#if USE_TEXTURE_ADDRESS
    // update the array to the texture
    hipMemcpyToArray(arr_mcv, 0, 0, d_mcv, ncheck * nmaxX2 * sizeof(int), hipMemcpyDeviceToDevice);
#endif

    // step 2: variable to check nodes
	updateVariableNode_gpu();

#if USE_TEXTURE_ADDRESS
    // update the array to the texture
    hipMemcpyToArray(arr_mvc, 0, 0, d_mvc, nvar * nmaxX1 * sizeof(int), hipMemcpyDeviceToDevice);
#endif

#if	USE_TABLE_CODE
	updateConstantMemoryLLRByte( d_LLRout );
#endif

	if (psc && syndrome_check_gpu()) {
	  is_valid_codeword = true;
      break;
    }
  }
  while (iter < max_iters);

  hipMemcpy( LLRout, d_LLRout, nvar * sizeof(int), hipMemcpyDeviceToHost );


  return (is_valid_codeword ? iter : -iter);
}

int ldpc_gpu::bp_decode_once(char *LLRout, int code_rate, 
	int *LLRin/*= NULL*/, 
	bool psc /*= true*/,			//!< check syndrom after each iteration
	int max_iters /*= 50*/ )		//!< Maximum number of iterations
{
	m_ldpcCurrent = m_ldpcDataPool.findLDPC_DATA( code_rate );
	nvar = m_ldpcCurrent->nvar;
	ncheck = m_ldpcCurrent->ncheck;

	if( LLRin )
	hipMemcpy( d_LLRin, LLRin, nvar * sizeof(int), hipMemcpyHostToDevice );

 	dim3 block( SIZE_BLOCK );
	dim3 grid( (nvar + block.x - 1) / block.x );

	// initial step
	initializeMVC_kernel<<< grid, block >>>( nvar, m_ldpcCurrent->d_sumX1, d_LLRin, m_ldpcCurrent->d_mvc, m_ldpcCurrent->nmaxX1, 1  );

#if WRITE_FILE_FOR_DRIVER
	static bool bRunOnce1 = false;
	if( !bRunOnce1 ){
		hipMemcpy( h_mvc, m_ldpcCurrent->d_mvc, nvar * m_ldpcCurrent->nmaxX1 * sizeof(int), hipMemcpyDeviceToHost );
		writeArray( h_mvc, nvar * m_ldpcCurrent->nmaxX1, "../data/mvcInit.txt" );
	
		hipMemcpy( h_LLRin, d_LLRin, nvar * sizeof(int), hipMemcpyDeviceToHost );
		writeArray( h_LLRin, nvar, "../data/input.txt" );
		bRunOnce1 = true;
	}
#endif

	int not_valid_codeword = true;
	int iter = 1;
	for( ; iter < max_iters && not_valid_codeword; iter ++ )
	{
		// --------- Step 1: check to variable nodes ----------
		updateCheckNodeOpti_kernel<<< grid, block >>>(ncheck, nvar, 
			m_ldpcCurrent->d_sumX2, m_ldpcCurrent->d_mvc,
			m_ldpcCurrent->d_jind, m_ldpcCurrent->d_logexp_table, 
			m_ldpcCurrent->Dint1, m_ldpcCurrent->Dint2, m_ldpcCurrent->Dint3,QLLR_MAX, 
			m_ldpcCurrent->nmaxX1, m_ldpcCurrent->nmaxX2, 1, 
			m_ldpcCurrent->d_mcv );	// Shared not faster

				
#if WRITE_FILE_FOR_DRIVER
		static bool bRunOnce1 = false;
		if( iter == 1 && !bRunOnce1 ){
			hipMemcpy( h_mcv, m_ldpcCurrent->d_mcv, ncheck * m_ldpcCurrent->nmaxX2 * sizeof(int), hipMemcpyDeviceToHost );

			writeArray( h_mcv, ncheck * m_ldpcCurrent->nmaxX2, "../data/mcv.txt" );

			bRunOnce1 = true;
		}
#endif

		// --------- Step 2: variable to check nodes ----------
#if USE_BLOCK_2D
	
	dim3 block( SIZE_BLOCK_2D_X, MAX_VAR_NODE );
	dim3 grid;
	grid.x = (nvar * MAX_VAR_NODE + SIZE_BLOCK_2D_X * MAX_VAR_NODE - 1) 
				/ (SIZE_BLOCK_2D_X * MAX_VAR_NODE) ;

	updateVariableNodeOpti2D_kernel<<< grid, block >>>( nvar, ncheck, 
		d_sumX1, d_mcv, d_iind, d_LLRin, 
		d_LLRout, d_mvc );
#else
		updateVariableNodeOpti_kernel<<< grid, block >>>( nvar, ncheck, 
			m_ldpcCurrent->d_sumX1, m_ldpcCurrent->d_mcv, m_ldpcCurrent->d_iind, d_LLRin, 
			m_ldpcCurrent->nmaxX1, m_ldpcCurrent->nmaxX2, 1, 
			d_LLRout, m_ldpcCurrent->d_mvc );
#endif

		// --------- Step 3: check syndrome ��żУ�� ----------
#if 0
		syndrome_check_kernel<<< grid, block >>>( d_LLRout, d_sumX2, ncheck, d_V, 
			d_synd );

		hipMemcpy( &not_valid_codeword, d_synd, sizeof(int), hipMemcpyDeviceToHost );
#else
		hipMemcpy( LLRout, d_LLRout, nvar * sizeof(char), hipMemcpyDeviceToHost );
		
#if WRITE_FILE_FOR_DRIVER
		static bool bRunOnce2 = false;
		if( iter == 1 && !bRunOnce2 ){
			hipMemcpy( h_mvc, m_ldpcCurrent->d_mvc, nvar * m_ldpcCurrent->nmaxX1 * sizeof(int), hipMemcpyDeviceToHost );

			writeArray( LLRout, nvar, "../data/output.txt" );
			writeArray( h_mvc, nvar * m_ldpcCurrent->nmaxX1, "../data/mvc.txt" );		

			bRunOnce2 = true;
		}
#endif

		if (psc && check_parity_cpu(LLRout)) {
			 not_valid_codeword = false;
			break;
		}
#endif
	}
  
  //hipMemcpy( LLRout, d_LLRout, nvar * sizeof(char), hipMemcpyDeviceToHost );


  return (!not_valid_codeword ? iter : -iter);
}

bool ldpc_gpu::check_parity_cpu(char *LLRN, int nFrame) 
{
	// Please note the IT++ convention that a sure zero corresponds to
	// LLR=+infinity
	int i, j, synd, vi;

	for( int frame = 0; frame < nFrame; frame ++ ) {
	char *LLR = LLRN + nvar * frame;

	for (j = 0; j < ncheck; j++) {
		synd = 0;
		int vind = j; // tracks j+i*ncheck
		for (i = 0; i < m_ldpcCurrent->h_sumX2[j]; i++) {
			vi = m_ldpcCurrent->h_V[vind];
			if (LLR[vi]) {
				synd++;
			}
			vind += ncheck;
		}
		if ((synd&1) == 1) {
			return false;  // codeword is invalid
		}
	}
	}
	return true;   // codeword is valid
}

bool ldpc_gpu::initialize( LDPC_CodeFactory* pcodes, scmplx* psymbols, int nMultiMax )
{
	m_nMultiMax = nMultiMax;

	m_ldpcDataPool.initialize( pcodes, nMultiMax );
	LDPC_DATA_GPU* m_ldpcCurrent = m_ldpcDataPool.findLDPC_DATA(0);
	nvar = m_ldpcCurrent->nvar;
	ncheck = m_ldpcCurrent->ncheck;

	//max_cnd = 200;
	QLLR_MAX = (1<<31 -1)>>4;//(std::numeric_limits<int>::max() >> 4);

	hipMalloc( (void**)&d_LLRin, m_nMultiMax * nvar * sizeof(int) );
	hipMalloc( (void**)&d_LLRout, m_nMultiMax * nvar * sizeof(char) );
	hipMemset( d_LLRout, 1, m_nMultiMax * nvar * sizeof(char) );

	hipMalloc( (void**)&d_synd, nMultiMax * sizeof(int) );
	hipMemset( d_synd, 0, nMultiMax * sizeof(int) );
	
	hipMalloc( (void**)&d_pSymbolsTemplate, M_SYMBOL_SIZE_MAX * M_CONST_NUMBER * sizeof(scmplx) );
	hipMemcpy( d_pSymbolsTemplate, psymbols, M_SYMBOL_SIZE_MAX * M_CONST_NUMBER * sizeof(scmplx), hipMemcpyHostToDevice );
	m_pSymbolsTemplate = psymbols;

	hipMalloc( (void**)&d_pSymbolsIn, m_nMultiMax * FRAME_SIZE_NORMAL * sizeof(scmplx) );

	hipMalloc( (void**)&d_pDist2, m_nMultiMax * M_SYMBOL_SIZE_MAX * FRAME_SIZE_NORMAL * sizeof(float) );

	hipMalloc( (void**)&d_pSoftBitCache, m_nMultiMax * FRAME_SIZE_NORMAL * sizeof(int) );

	m_pDist2 = new float[m_nMultiMax*FRAME_SIZE_NORMAL*M_SYMBOL_SIZE_MAX];

	initConstantMemoryLogExp(m_ldpcCurrent->getCode()->llrcalc.logexp_table._data());

#if USE_TEXTURE_ADDRESS
	// cuda texture ------------------------------------------------------------------------------------------
	channelDesc = hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindSigned);
    hipError_t err = hipMallocArray(&arr_mcv, &channelDesc, ncheck, nmaxX2);
    hipMemcpyToArray(arr_mcv, 0, 0, d_mcv, ncheck * nmaxX2 * sizeof(int), hipMemcpyDeviceToDevice);

	texMCV.addressMode[0] = hipAddressModeClamp;
	texMCV.addressMode[1] = hipAddressModeClamp;
    texMCV.filterMode = hipFilterModePoint;
    texMCV.normalized = false;

	hipBindTextureToArray(texMCV, arr_mcv, channelDesc);

	hipMallocArray(&arr_mvc, &channelDesc, nvar, nmaxX1);
    hipMemcpyToArray(arr_mvc, 0, 0, d_mvc, nvar * nmaxX1 * sizeof(int), hipMemcpyDeviceToDevice);
	hipBindTextureToArray(texMVC, arr_mvc, channelDesc);

#endif

	h_mvc = (int*)malloc(nvar * MAX_LOCAL_CACHE * sizeof(int));
	h_mcv = (int*)malloc(ncheck * MAX_LOCAL_CACHE * sizeof(int));
	h_LLRin= (int*)malloc(nvar * sizeof(int));
	h_pSoftBit = (int*)malloc(m_nMultiMax * FRAME_SIZE_NORMAL* sizeof(int));

	return true;
}


void ldpc_gpu::release()
{
	if( NULL == d_LLRin )
		return;

	hipFree( d_LLRin );	hipFree( d_LLRout );
	d_LLRin = NULL;			d_LLRout = NULL;

	hipFree( d_synd );
	d_synd = NULL;

	hipFree( d_pSymbolsTemplate );
	d_pSymbolsTemplate = NULL;

	hipFree( d_pDist2 );	d_pDist2 = NULL;

	free( h_mvc );	free( h_mcv );	free( h_LLRin );
	h_mvc = NULL;	h_mcv = NULL;	h_LLRin = NULL;
	free( m_pDist2 );
	m_pDist2 = NULL;

	return ;
}

ldpc_gpu::~ldpc_gpu()
{
	release();
}

int ldpc_gpu::bp_decode_once( itpp::vec& softbits, char *LLRout, int code_rate )
{
	m_ldpcCurrent = m_ldpcDataPool.findLDPC_DATA( code_rate );
	nvar = m_ldpcCurrent->nvar;
	ncheck = m_ldpcCurrent->ncheck;
	itpp::QLLRvec llrIn = m_ldpcCurrent->getCode()->get_llrcalc().to_qllr(softbits);

	return bp_decode_once( LLRout, code_rate, llrIn._data() );	
}

int ldpc_gpu::bp_decode_once( double* softbits, char *LLRout, int code_rate )
{
	m_ldpcCurrent = m_ldpcDataPool.findLDPC_DATA( code_rate );
	nvar = m_ldpcCurrent->nvar;
	ncheck = m_ldpcCurrent->ncheck;
	itpp::vec  softVec( nvar );
	convertBufferToVec( softbits, softVec );
	return bp_decode_once( softVec, LLRout, code_rate );
}

float ldpc_gpu::distance( const scmplx& cL, const scmplx& cR )
{
	float dist2 = 0;
	dist2 = (float)(cL.im - cR.im) * (cL.im - cR.im) + (float)(cL.re - cR.re) * (cL.re - cR.re);
	return dist2;
}

int ldpc_gpu::decode_soft( scmplx* sym, double N0, int nPayloadSymbols, int M, int k,
	int *pFrame, int code_rate,
	double* p_soft_bits, double* p_soft_bits_cache,
	char* p_bitLDPC, int nMulti /*= 1*/ )
{
	m_ldpcCurrent = m_ldpcDataPool.findLDPC_DATA( code_rate );
	nvar = m_ldpcCurrent->nvar;
	ncheck = m_ldpcCurrent->ncheck;

	for ( int j = 0;j<nMulti;j++ )
		hipMemcpy( d_pSymbolsIn+j*nPayloadSymbols, 
			sym + j*FRAME_SIZE_NORMAL, 
			nPayloadSymbols * sizeof(scmplx), 
			hipMemcpyHostToDevice );// 0.02 ms/1f, 0.28 ms/3f

	dim3 block( 1024/M );
	dim3 grid;
	grid.x = (nPayloadSymbols + block.x-1)/block.x;

	block.y = M;
	distance_kernel<<< grid, block >>>( d_pSymbolsIn, d_pSymbolsTemplate+(k-2)*M_SYMBOL_SIZE_MAX, M, d_pDist2, CP, nPayloadSymbols, nMulti );// 0.13 ms/1f, 0.28 ms/3f

	block.y = k;
	soft_bit_kernel<<< grid, block >>>(d_pDist2, d_pSoftBitCache, k, M, N0, 
		m_ldpcCurrent->Dint1, QLLR_MAX, nPayloadSymbols, nMulti );// 0.1 ms/1f, 0.23 ms/3f

	
#if WRITE_FILE_FOR_DRIVER
	static bool bRunOnce1 = false;
	if( !bRunOnce1 ){
		std::vector<int*> params;
		params.push_back(&k);
		params.push_back(&M);
		int Dint1 = m_ldpcCurrent->Dint1;
		params.push_back(&Dint1);
		params.push_back(&QLLR_MAX);
		params.push_back(&nPayloadSymbols);
		params.push_back(&nMulti);
		writeFile( params, "../data/softBitSize.txt" );

		hipMemcpy( m_pDist2, d_pDist2, nMulti * M_SYMBOL_SIZE_MAX * FRAME_SIZE_NORMAL * sizeof(float), hipMemcpyDeviceToHost );
		writeArray( m_pDist2, nMulti * M_SYMBOL_SIZE_MAX * FRAME_SIZE_NORMAL, "../data/pDist2.txt" );
	
		hipMemcpy( h_pSoftBit, d_pSoftBitCache, nMulti * FRAME_SIZE_NORMAL * sizeof(int), hipMemcpyDeviceToHost );
		writeArray( h_pSoftBit, nMulti * FRAME_SIZE_NORMAL, "../data/pSoftBit.txt" );
		bRunOnce1 = true;
	}
#endif

	// step	2:	de-interleave
#if 1
	reorder_kernel<<< grid, block >>>(d_LLRin, d_pSoftBitCache, k, nPayloadSymbols, nMulti );// 0.02 ms/1f, 0.03 ms/3f
	//hipMemcpy( p_soft_bits, d_LLRin, FRAME_SIZE_NORMAL*sizeof(int), hipMemcpyDeviceToHost );
#else
	hipMemcpy( p_soft_bits_cache, d_pSoftBitCache, FRAME_SIZE_NORMAL*sizeof(int), hipMemcpyDeviceToHost );
	for( int i = 0; i < nPayloadSymbols; i++ )
		for (int j=0;j<k;j++)
			p_soft_bits[j*nPayloadSymbols+i] = p_soft_bits_cache[i*k+j];	
#endif

	// step	3:	ldpc decode
	bp_decode_once( p_bitLDPC, code_rate, nMulti );// 1.8 ms/1f, 5.0 ms/3f

	// step	4:	cast type, char -> int
	for( int frame = 0; frame < nMulti; frame ++ ) {
	for( int i = 0; i < nvar; i++ )
		pFrame[i+frame*FRAME_SIZE_NORMAL] = p_bitLDPC[i+frame*nvar];

	}

	hipError_t ce = hipGetLastError();
	if( hipSuccess != ce )
		cout << "bp_decode_once hipGetLastError = " << ce << endl;

	return 0;
}

void	ldpc_gpu::updateSymbolsTemplate( scmplx* psymbols )
{
	hipMemcpy( d_pSymbolsTemplate, psymbols, M_SYMBOL_SIZE_MAX * M_CONST_NUMBER * sizeof(scmplx), hipMemcpyHostToDevice );
}

int ldpc_gpu::bp_decode_once(char *LLRout, int code_rate, int nMulti, 
	int *LLRin/*= NULL*/, 
	bool psc /*= true*/,			//!< check syndrom after each iteration
	int max_iters /*= 50*/ )		//!< Maximum number of iterations
{
	StopWatchInterface	*timerStep;

	sdkCreateTimer( &timerStep );

	m_ldpcCurrent = m_ldpcDataPool.findLDPC_DATA( code_rate );
	nvar = m_ldpcCurrent->nvar;
	ncheck = m_ldpcCurrent->ncheck;

	if( LLRin )
	hipMemcpy( d_LLRin, LLRin, nvar * sizeof(int), hipMemcpyHostToDevice );

 	dim3 block( SIZE_BLOCK );
	dim3 grid( (nvar + block.x - 1) / block.x );

	// initial step 0.04ms
	initializeMVC_kernel<<< grid, block >>>( nvar, m_ldpcCurrent->d_sumX1, d_LLRin, m_ldpcCurrent->d_mvc, m_ldpcCurrent->nmaxX1, nMulti );

	int not_valid_codeword = true;
	int iter = 1;
	for( ; iter < max_iters && not_valid_codeword; iter ++ )
	{
		// --------- Step 1: check to variable nodes ----------	0.31 ms
		updateCheckNodeOpti_kernel<<< grid, block >>>(ncheck, nvar, 
			m_ldpcCurrent->d_sumX2, m_ldpcCurrent->d_mvc,
			m_ldpcCurrent->d_jind, m_ldpcCurrent->d_logexp_table, 
			m_ldpcCurrent->Dint1, m_ldpcCurrent->Dint2, m_ldpcCurrent->Dint3,QLLR_MAX, 
			m_ldpcCurrent->nmaxX1, m_ldpcCurrent->nmaxX2, nMulti, 
			m_ldpcCurrent->d_mcv );	// Shared not faster


		// --------- Step 2: variable to check nodes ----------	0.23 ms
		updateVariableNodeOpti_kernel<<< grid, block >>>( nvar, ncheck, 
			m_ldpcCurrent->d_sumX1, m_ldpcCurrent->d_mcv, m_ldpcCurrent->d_iind, d_LLRin, 
			m_ldpcCurrent->nmaxX1, m_ldpcCurrent->nmaxX2, nMulti, 
			d_LLRout, m_ldpcCurrent->d_mvc );


		// --------- Step 3: check syndrome ��żУ�� ---------- 0.11 ms	
#if ENABLE_TIME_TEST
		hipDeviceSynchronize();
		sdkStartTimer( &timerStep );
#endif
		bool bVaid = syndrome_check_gpu( nMulti );// 0.17 ms 
		
#if ENABLE_TIME_TEST
		sdkStopTimer( &timerStep );
		cout  << "timerStepValue = "<< sdkGetTimerValue( &timerStep ) << " ms, " << endl;
#endif
	
		if (psc && bVaid ) {
			 not_valid_codeword = false;
			break;
		}

	}

	hipMemcpy( LLRout, d_LLRout, nMulti * nvar * sizeof(char), hipMemcpyDeviceToHost );

  return (!not_valid_codeword ? iter : -iter);
}

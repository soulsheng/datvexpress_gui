

#include "bch_bm_decode.cuh"
#include "bch_bm_decode_kernel.cuh"
#include <hip/hip_runtime.h>

#include <vector>
#include <stdlib.h>
#include <iostream>
using namespace std;

#include "dvbUtility.h"

#if USE_TEXTURE_ADDRESS
	hipArray* arr_alpha;
	hipChannelFormatDesc channelDesc;
#endif

bch_gpu::bch_gpu()
{

}

bch_gpu::~bch_gpu()
{

}


void bch_gpu::initialize(	int *powAlpha, int *indexAlpha, int mNormal, 
							int *S, int nS, 
							int n, int tCapacity, int MAXN, int tMax, int nMultiMax )
{
	m_nAlphaSize = 1<<mNormal;
	m_nSSize = nS;
	this->n = n;
	this->tCapacity = tCapacity;
	this->tMax = tMax;

	hipMalloc( (void**)&d_powAlpha, m_nAlphaSize*sizeof(int) );
	hipMalloc( (void**)&d_indexAlpha, m_nAlphaSize*sizeof(int) );
	hipMalloc( (void**)&d_S, m_nSSize*sizeof(int) );

	hipMemcpy( d_powAlpha, powAlpha, m_nAlphaSize*sizeof(int), hipMemcpyHostToDevice );
	hipMemcpy( d_indexAlpha, indexAlpha, m_nAlphaSize*sizeof(int), hipMemcpyHostToDevice );
	hipMemcpy( d_S, S, nS*sizeof(int), hipMemcpyHostToDevice );

	hipMalloc( (void**)&d_codeword, nMultiMax * n*sizeof(char) );
	
	
	hipMalloc( (void**)&d_SCache, nMultiMax * tCapacity*2*BLOCK_DIM*sizeof(int) );
	hipMemset( d_SCache, 0, nMultiMax * tCapacity*2*BLOCK_DIM*sizeof(int) );

	hipMalloc( (void**)&d_lambda, nMultiMax* tCapacity*2*sizeof(int));
	
	hipMalloc( (void**)&d_el, nMultiMax* tMax*2*sizeof(int));
	hipMemset( d_el, -1, nMultiMax* tMax*2*sizeof(int) );
	
	hipMalloc( (void**)&d_kk, nMultiMax*sizeof(int));
	hipMemset( d_kk, 0, nMultiMax*sizeof(int) );

	hipMalloc( (void**)&d_L, nMultiMax * sizeof(int)  );
	hipMemset( d_L, 0, nMultiMax*sizeof(int) );

	m_SCache = (int*) calloc(nMultiMax * tCapacity*2*BLOCK_NUM_MAX,sizeof(int));

	this->powAlpha = powAlpha;
	this->indexAlpha = indexAlpha;
	this->S = S;
	this->MAXN = MAXN;

	
#if USE_TEXTURE_ADDRESS
	// cuda texture ------------------------------------------------------------------------------------------
	channelDesc = hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindSigned);
    hipError_t err = hipMallocArray(&arr_alpha, &channelDesc, m_nAlphaSize, 1);
    hipMemcpyToArray(arr_alpha, 0, 0, d_powAlpha, m_nAlphaSize * sizeof(int), hipMemcpyDeviceToDevice);

	texAlpha.addressMode[0] = hipAddressModeClamp;
    texAlpha.filterMode = hipFilterModePoint;
    texAlpha.normalized = false;

	hipBindTextureToArray(texAlpha, arr_alpha, channelDesc);

#endif
}

void bch_gpu::release()
{
	hipFree( d_powAlpha );
	hipFree( d_indexAlpha );
	hipFree( d_S );
	hipFree( d_SCache );

	hipFree( d_codeword );

	hipFree( d_lambda ); 
	hipFree( d_el );
	hipFree( d_kk );

	hipFree( d_L );
#if USE_TEXTURE_ADDRESS
	//hipFreeArray( arr_alpha );
#endif
}


bool bch_gpu::error_detection( char* codeword, int nMulti )
{
	this->codeword = codeword;
	hipMemcpy( d_codeword, codeword, nMulti * n*sizeof(char), hipMemcpyHostToDevice );

	dim3 block(BLOCK_DIM);
	dim3 grid( (n+BLOCK_DIM-1)/BLOCK_DIM );

#if 1

	error_detection_kernel<<< grid, block >>>( d_codeword, d_powAlpha, d_SCache, char(tCapacity*2), MAXN, n, nMulti );

#else
	for(int i = 0; i < tCapacity*2; i++)
	{
		error_detection_kernel<<< grid, block >>>( d_codeword, d_powAlpha, d_SCache, i, MAXN, n );
	}
#endif

	hipMemcpy( m_SCache, d_SCache, nMulti * tCapacity*2*grid.x * sizeof(int), hipMemcpyDeviceToHost );
		
		
#if WRITE_FILE_FOR_DRIVER
	static bool bRunOnce1 = false;
	if( !bRunOnce1 ){
		writeFile( n, m_nAlphaSize, grid.x, MAXN, "../data/bchSize.txt" );
		writeArray( codeword, n, "../data/codeword.txt" );		
		writeArray( powAlpha, m_nAlphaSize, "../data/powAlpha.txt" );
		writeArray( m_SCache, grid.x, "../data/SCache.txt" );

		bRunOnce1 = true;
	}
#endif

	bool syn = false;
	for( int frame = 0; frame < nMulti; frame ++ )	{

	for(int i = 0; i < tCapacity*2; i++)
	{
		S[i+frame*tCapacity*2] = 0;
		for( int j=0; j< grid.x; j++ )
		{
			S[i+frame*tCapacity*2] ^= m_SCache[j+i*grid.x+frame*tCapacity*2*grid.x];
		}
	}
	
	for(int i = 0; i < tCapacity*2; i++)
	{
		S[i+frame*tCapacity*2] = indexAlpha[S[i+frame*tCapacity*2]];

		if(S[i+frame*tCapacity*2] != -1)
			syn = true;

	}
	}

	return syn;
}

void bch_gpu::chienSearch( int* lambda, int* el, int* L, int nMulti )
{

	// 0.6 ms 
	hipMemcpy( d_lambda, lambda, nMulti * tCapacity * 2 * sizeof(int), hipMemcpyHostToDevice );
	hipMemcpy( d_L, L, nMulti*sizeof(int), hipMemcpyHostToDevice );

	hipMemset( d_el, -1, nMulti * tMax*2*sizeof(int) );
	hipMemset( d_kk, 0, nMulti*sizeof(int) );

	dim3 block( BLOCK_DIM );
	dim3 grid( (MAXN + BLOCK_DIM - 1)/BLOCK_DIM );
	chien_search_kernel<<< grid, block >>>( d_powAlpha, d_lambda, d_el, d_kk, d_L, MAXN, 
		tMax, tCapacity * 2, nMulti );

	hipMemcpy( el, d_el, nMulti * tMax * 2 * sizeof(int), hipMemcpyDeviceToHost );

#if WRITE_FILE_FOR_DRIVER
	static bool bRunOnce1 = false;
	if( !bRunOnce1 ){
		std::vector<int> paramSize;
		paramSize.push_back( tCapacity );
		paramSize.push_back( m_nAlphaSize );
		paramSize.push_back( tMax );
		paramSize.push_back( MAXN );
		paramSize.push_back( L );
		//writeFile( tCapacity, m_nAlpha, tMax, MAXN, L, "../data/chienSearchSize.txt" );
		writeFile( paramSize, "../data/chienSearchSize.txt" );
		writeArray( lambda, tCapacity * 2, "../data/lambda.txt" );		
		writeArray( powAlpha, m_nAlphaSize, "../data/powAlpha.txt" );
		writeArray( el, tMax * 2, "../data/el.txt" );

		bRunOnce1 = true;
	}
#endif
}
